#include "hip/hip_runtime.h"
/*covargpu0.cu*/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include "setup.h"
#include "covargpu0.h"
#include "newdcdio.h"

__global__ void covargpu0_add(float *covar_vals, float *X, float *Y, float *Z, int N, int num_frames){

	float delX_i, delY_i, delZ_i;
	float delX_j, delY_j, delZ_j;

	float numerator, denominator;
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	while(tidx < N){
		int tidy = threadIdx.y + blockIdx.y * blockDim.y;
		while(tidy < N){
        	for (int frame = 0; frame < num_frames; frame++) {
				if(tidx <= tidy){					
					int index1 = frame*N+tidx, index2 = frame*N+tidy;
					delX_i = X[index1]-X[tidx];
					delY_i = Y[index1]-Y[tidx];
					delZ_i = Z[index1]-Z[tidx];
					delX_j = X[index2]-X[tidy];
					delY_j = Y[index2]-Y[tidy];
					delZ_j = Z[index2]-Z[tidy];
					if((delX_i == 0 && delY_i == 0 && delZ_i == 0)||(delX_j == 0 && delY_j == 0 && delZ_j == 0)){
						covar_vals[tidx*N + tidy] += 0;
					}
					else{
						numerator = delX_i*delX_j + delY_i*delY_j + delZ_i*delZ_j;
						denominator = sqrtf((delX_i*delX_i + delY_i*delY_i + delZ_i*delZ_i)*(delX_j*delX_j + delY_j*delY_j + delZ_j*delZ_j));
						covar_vals[tidx*N + tidy] += numerator/denominator;
						if(tidx!=tidy) covar_vals[tidy*N + tidx] += numerator/denominator;
					}
				}
			}
			tidy += blockDim.y * gridDim.y;
		}
		tidx += blockDim.x * gridDim.x;
	}
	return;
}



extern "C" int covargpu0_setup(setup_data *setup){
	//Get GPU from user
	int dev;
	printf("Enter GPU ID: "); fflush(stdout);
	scanf("%d", &dev);
	hipSetDevice(dev);
	//printf("Setting up device memory\n");

	//Allocate space for linearized XYZ arrays
	int num_frames = setup->end_frame-setup->start_frame+1;	
	hipMalloc((void**)&setup->dev_X, num_frames*sizeof(float)*setup->N);
	hipMalloc((void**)&setup->dev_Y, num_frames*sizeof(float)*setup->N);
	hipMalloc((void**)&setup->dev_Z, num_frames*sizeof(float)*setup->N); 
	//printf("XYZ allocated\n");

	sprintf(setup->dcd_filename, "%s/%s/dcd/%s_%d_%s.dcd", setup->protein_name, setup->sim_type, setup->protein_name, setup->runstart, setup->sim_type);
	FILE *dcd_file = fopen(setup->dcd_filename, "r");
	if(dcd_file == NULL){
		printf("Failed to open file: %s\n", setup->dcd_filename);
		exit(0);
	}
	float tempXYZ[setup->N*3];
	int iin;
	fread(&iin, 4, 1, dcd_file);
	//printf("%d\n", iin);
	fread(tempXYZ, 4*setup->N, 1, dcd_file);
	fread(&iin, 4, 1, dcd_file);
	//printf("%d\n", iin);
	fread(&iin, 4, 1, dcd_file);
	//printf("%d\n", iin);
	fread(&tempXYZ[setup->N], 4*setup->N, 1, dcd_file);
	fread(&iin, 4, 1, dcd_file);
	//printf("%d\n", iin);
	fread(&iin, 4, 1, dcd_file);
	//printf("%d\n", iin);
	fread(&tempXYZ[setup->N*2], 4*setup->N, 1, dcd_file);
	fread(&iin, 4, 1, dcd_file);
	//printf("%d\n", iin);
	fclose(dcd_file);

	//Allocate host covar vals, initialize to 0
	setup->lin_covar_vals = (float *) malloc(setup->N*setup->N*sizeof(float));
	for(int i = 0; i < setup->N*setup->N; i++) setup->lin_covar_vals[i] = 0;

	//Allocate device covar vals, copy 0 initialized matrix
	hipMalloc((void**)&setup->dev_covar_vals, setup->N*setup->N*sizeof(float*));
	hipMemcpy(setup->dev_covar_vals, setup->lin_covar_vals, setup->N*setup->N*sizeof(float), hipMemcpyHostToDevice);


	//printf("Allocation complete\n");
	return 0;
}


extern "C" int covargpu0(setup_data *setup, float ***X, float ***Y, float ***Z){
	//Allocate linearized, contiguous arrays for XYZ (num_frames x N)
	int num_frames = setup->end_frame-setup->start_frame+1;	
	float *linX = (float*) malloc(setup->N*num_frames*sizeof(float));	
	float *linY = (float*) malloc(setup->N*num_frames*sizeof(float));
	float *linZ = (float*) malloc(setup->N*num_frames*sizeof(float));
	//printf("lin declared\n");
	for(int i = setup->start_frame; i <= setup->end_frame; i++){
		for(int j = 0; j < setup->N; j++){
			linX[setup->N*(i-setup->start_frame) + j] = (*X)[i][j];
			linY[setup->N*(i-setup->start_frame) + j] = (*Y)[i][j];
			linZ[setup->N*(i-setup->start_frame) + j] = (*Z)[i][j];
		}
	}	//printf("Linear matrices created\n");


	//Copy XYZ coordinate arrays to device
	hipMemcpy(setup->dev_X, linX, setup->N*num_frames*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(setup->dev_Y, linY, setup->N*num_frames*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(setup->dev_Z, linZ, setup->N*num_frames*sizeof(float), hipMemcpyHostToDevice);
	//printf("XYZ copied\n");

	//for(int i = 0; i < setup->N*num_frames; i++) printf("%8.3f%8.3f%8.3f\n", linX[i], linY[i], linZ[i]);

	//setup timing variables
	hipEvent_t start, stop;
	float elapsed_time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//block and thread dimension setup
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(32, 32);

	//Call kernel to add to running sum
	covargpu0_add<<<numBlocks,threadsPerBlock>>>(setup->dev_covar_vals, setup->dev_X, setup->dev_Y, setup->dev_Z, setup->N, num_frames);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("Covariance calculated in %3.2f s\n", elapsed_time/1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	//Free host linear arrays
	free(linX);
	free(linY);
	free(linZ);

	return 0;
}



extern "C" int covargpu0_post(setup_data *setup){

	//Create/Open .dat file
	char dat_filename[50];
	FILE *dat_file;
	sprintf(dat_filename, "%s/%s/dat/covargpu0_%s_%d-%d_%d-%d.dat", setup->protein_name, setup->sim_type, setup->protein_name, setup->start_frame, setup->end_frame, setup->runstart, setup->runstart+setup->runnum);
	dat_file = fopen(dat_filename, "w");
	if(dat_file == NULL){
		printf("Failed to open file: %s", dat_filename);
		exit(0);
	}

	//Copy running sum back to host
	hipMemcpy(setup->lin_covar_vals, setup->dev_covar_vals, setup->N*setup->N*sizeof(float), hipMemcpyDeviceToHost);

	//Print matrix to .dat file
	int iterations = (setup->runcount*(setup->end_frame-setup->start_frame));	
	for(int i = 0; i < setup->N; i++){
		for(int j = 0; j < setup->N; j++){
			fprintf(dat_file, "%10.6f", setup->lin_covar_vals[i*setup->N + j]/iterations);
		}
	fprintf(dat_file, "\n");
	}
	fclose(dat_file);

	//Free memory
	hipFree(setup->dev_X);
	hipFree(setup->dev_Y);
	hipFree(setup->dev_Z);
	hipFree(setup->dev_covar_vals);
	free(setup->lin_covar_vals);
	return 0;
}
